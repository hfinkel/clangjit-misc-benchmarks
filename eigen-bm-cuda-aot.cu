#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <chrono>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include <Eigen/Core>

// A benchmark for ClangJIT in the style of:
// https://github.com/eigenteam/eigen-git-mirror/blob/master/bench/benchmark.cpp

using namespace std;
using namespace Eigen;

#ifndef __has_feature
  #define __has_feature(x) 0
#endif

template <typename T>
__global__ void dokernel(int repeat, int size, T *out) {
  Matrix<T,Dynamic,Dynamic> I = Matrix<T,Dynamic,Dynamic>::Ones(size, size);
  Matrix<T,Dynamic,Dynamic> m(size, size);
  for(int i = 0; i < size; i++)
  for(int j = 0; j < size; j++) {
    m(i,j) = (i+size*j);
  }

  for (int r = 0; r < repeat; ++r) {
    m = Matrix<T,Dynamic,Dynamic>::Ones(size, size) + T(0.00005) * (m + (m*m));
  }

  out[threadIdx.x] = m(0, 0);
}

template <typename T>
void test_aot_i(int size, int repeat) {
  auto start = chrono::system_clock::now();

  cudaThreadSetLimit(hipLimitMallocHeapSize, 128*1024*1024);

  const int w = 1;
  T *harr = new T[w];
  memset(harr, 0, sizeof(T)*w);
  T *darr;
  hipMalloc((void **) &darr, w*sizeof(T));
  hipMemcpy(darr, harr, w*sizeof(T), hipMemcpyHostToDevice);

  dokernel<T><<<1, w>>>(repeat, size, darr);

  hipMemcpy(harr, darr, w*sizeof(T), hipMemcpyDeviceToHost);

#if 0
  for (int i = 0; i < w; ++i)
    cout << "h" << i << ": " << harr[i] << "\n";
#endif

  auto end = chrono::system_clock::now();
  cout << "JIT: " << std::chrono::duration<double>(end - start).count() << " s\n";
}

void test_aot(std::string &type, int size, int repeat) {
  if (type == "float")
    test_aot_i<float>(size, repeat);
  else if (type == "double")
    test_aot_i<double>(size, repeat);
  else
    cout << type << "not supported for AoT\n";
}

int main(int argc, char *argv[]) {
  int repeat = 40000000;
  if (argc > 1)
    repeat = atoi(argv[1]);

  int size = 3;
  if (argc > 2)
    size = atoi(argv[2]);

  string type("double");
  if (argc > 3)
    type = argv[3];

  test_aot(type, size, repeat);

  return 0;
}

